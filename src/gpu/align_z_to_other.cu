#include "hip/hip_runtime.h"
//
// Created by konrad on 10/23/17.
//

#define HAVE_CUDA

#include "../../include/librealsense/gpu/align_z_to_other_helpers.h"
#include "../../include/librealsense/rs.h"
#include "../types.h"
#include <ros/console.h>
#include <array>
#include <cmath>

#if !defined (HAVE_CUDA) || defined (CUDA_DISABLER)
namespace gpu {
bool align_z_to_other(rsimpl::byte * z_aligned_to_other, const uint16_t * z_pixels, float z_scale, const rs_intrinsics & z_intrin, const rs_extrinsics & z_to_other, const rs_intrinsics & other_intrin) {
    ROS_WARN("GPU DISABLED");
    return false;
}
}

#else
namespace gpu {

    void __global__ cast_to_ushort(unsigned short * output, unsigned int * input, const rs_intrinsics intrin) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= intrin.width || y >= intrin.height) {
            return;
        }

        int pixel_index = y * intrin.width + x;
        output[pixel_index] = static_cast<unsigned short>(input[pixel_index]);

    }

    __device__ inline float get_depth(int z_pixel_index, float z_scale, unsigned short * z_pixels_gpu) { return z_scale * z_pixels_gpu[z_pixel_index]; }

    __device__ inline void transfer_pixel(int z_pixel_index, int other_pixel_index, unsigned short * z_pixels_gpu, unsigned int * out_z_gpu) {
        if(0 != atomicCAS(&(out_z_gpu[other_pixel_index]), 0, (unsigned int) z_pixels_gpu[z_pixel_index])) {
            atomicMin(&(out_z_gpu[other_pixel_index]), (unsigned int) z_pixels_gpu[z_pixel_index]);
        }
    }

void __global__ align_images(const rs_intrinsics depth_intrin, const rs_extrinsics depth_to_other,
                      const rs_intrinsics other_intrin, float z_scale, unsigned short * z_pixels_gpu, unsigned int * out_pixels_gpu) {



        float inv_fx = 1.0 / depth_intrin.fx;
        float inv_fy = 1.0 / depth_intrin.fy;
        // Iterate over the pixels of the depth image

        int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
        int depth_y = blockIdx.y * blockDim.y + threadIdx.y;

        if (depth_x >= depth_intrin.width || depth_y >= depth_intrin.height) {
            return;
        }

        int depth_pixel_index = depth_y * depth_intrin.width + depth_x;

        // Skip over depth pixels with the value of zero, we have no depth data so we will not write anything into our aligned images
        if (float depth = get_depth(depth_pixel_index, z_scale, z_pixels_gpu)) {
            // Map the top-left corner of the depth pixel onto the other image
            float depth_pixel[2] = {depth_x - 0.5f,
                                    depth_y - 0.5f}, depth_point[3], other_point[3], other_pixel[2];
            rs_deproject_pixel_to_point(depth_point, &depth_intrin, depth_pixel, depth, inv_fx, inv_fy);
            rs_transform_point_to_point(other_point, &depth_to_other, depth_point);
            rs_project_point_to_pixel(other_pixel, &other_intrin, other_point);
            const int other_x0 = static_cast<int>(other_pixel[0] + 0.5f);
            const int other_y0 = static_cast<int>(other_pixel[1] + 0.5f);

            // Map the bottom-right corner of the depth pixel onto the other image
            depth_pixel[0] = depth_x + 0.5f;
            depth_pixel[1] = depth_y + 0.5f;
            rs_deproject_pixel_to_point(depth_point, &depth_intrin, depth_pixel, depth, inv_fx, inv_fy);
            rs_transform_point_to_point(other_point, &depth_to_other, depth_point);
            rs_project_point_to_pixel(other_pixel, &other_intrin, other_point);
            const int other_x1 = static_cast<int>(other_pixel[0] + 0.5f);
            const int other_y1 = static_cast<int>(other_pixel[1] + 0.5f);

            if (other_x0 < 0 || other_y0 < 0 || other_x1 >= other_intrin.width ||
                other_y1 >= other_intrin.height)
                return;
            // Transfer between the depth pixels and the pixels inside the rectangle on the other image
            for (int y = other_y0; y <= other_y1; ++y)
                for (int x = other_x0; x <= other_x1; ++x)
                    transfer_pixel(depth_pixel_index, y * other_intrin.width + x, z_pixels_gpu, out_pixels_gpu);
        }
    }


    class CudaStreamWrapper {
    private:
        hipStream_t stream;
    public:
        CudaStreamWrapper() {
            hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
            ROS_INFO_THROTTLE(60, "GPU Stream Created");

        }

        ~CudaStreamWrapper() {
            hipStreamDestroy(stream);
            ROS_INFO_THROTTLE(60, "GPU Stream Destroyed");

        }

        operator hipStream_t&(){
            return stream;
        }


    };

    __host__ bool align_z_to_other(rsimpl::byte * z_aligned_to_other, const uint16_t * z_pixels, float z_scale, const rs_intrinsics & z_intrin, const rs_extrinsics & z_to_other, const rs_intrinsics & other_intrin)
    {
        ROS_INFO_THROTTLE(60, "GPU NOT DISABLED");

        int deviceCount = 0;
        auto cudaCallErrorStatus = hipGetDeviceCount(&deviceCount);
        if (hipSuccess != cudaCallErrorStatus) {
            ROS_ERROR("Failed to obtain number of GPUs. %s %s", hipGetErrorName(cudaCallErrorStatus), hipGetErrorString(cudaCallErrorStatus));
            return false;
        }
        if (deviceCount == 0) {
            ROS_WARN("NO ENABLED GPUs PRESENT");
            return false;
        }


        // ignore this error. technically the blocking sync can only be set once prior to device initialization. once
        // that initial setting is done, subsequent settings result in an error until device is restarted
        cudaCallErrorStatus = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

        auto out_z = (uint16_t *)(z_aligned_to_other);

        auto cudaDeleterUnsignedInt = [](unsigned int *ptr) {ROS_INFO_THROTTLE(60, "CUDA FREE INVOKED"); hipFree(ptr);};
        auto cudaDeleterUInt16t = [](uint16_t *ptr) {ROS_INFO_THROTTLE(60,"CUDA FREE INVOKED"); hipFree(ptr);};
        CudaStreamWrapper myCudaStream;
        // Kernel invocation
        dim3 threadsPerBlock(32, 32);
        dim3 numBlocks(static_cast<int> (std::ceil(static_cast<float>(z_intrin.width) / threadsPerBlock.x)),
                       static_cast<int>(std::ceil(
                static_cast<float>(z_intrin.height) / threadsPerBlock.y)));
        uint16_t * z_pixels_gpu;
        unsigned int * temporary_out_pixels_gpu;
        uint16_t * out_pixels_gpu_uint16t;


        cudaCallErrorStatus = hipMalloc((void **)&z_pixels_gpu, sizeof(uint16_t) * z_intrin.width * z_intrin.height);
        std::unique_ptr<uint16_t, decltype(cudaDeleterUInt16t)> z_pixels_gpu_u_ptr(z_pixels_gpu, cudaDeleterUInt16t);
        if (hipSuccess != cudaCallErrorStatus || deviceCount == 0) {
            ROS_ERROR("GPU processing failed while allocating space for z_pixels_gpu %s %s", hipGetErrorName(cudaCallErrorStatus), hipGetErrorString(cudaCallErrorStatus));
            return false;
        }
        cudaCallErrorStatus = hipMalloc((void **)&temporary_out_pixels_gpu, sizeof(unsigned int) * other_intrin.width * other_intrin.height);
        std::unique_ptr<unsigned int, decltype(cudaDeleterUnsignedInt)> out_pixels_gpu_u_ptr(temporary_out_pixels_gpu, cudaDeleterUnsignedInt);
        if (hipSuccess != cudaCallErrorStatus || deviceCount == 0) {
            ROS_ERROR("GPU processing failed while allocating space for temporary_out_pixels_gpu %s %s", hipGetErrorName(cudaCallErrorStatus), hipGetErrorString(cudaCallErrorStatus));
            return false;
        }
        cudaCallErrorStatus = hipMemsetAsync(temporary_out_pixels_gpu, sizeof(unsigned int) * other_intrin.width * other_intrin.height, 0, myCudaStream);
        if (hipSuccess != cudaCallErrorStatus) {
            ROS_ERROR("GPU processing failed while memsetting space for temporary_out_pixels_gpu %s %s", hipGetErrorName(cudaCallErrorStatus), hipGetErrorString(cudaCallErrorStatus));
            return false;
        }

        cudaCallErrorStatus = hipMalloc((void **)&out_pixels_gpu_uint16t, sizeof(uint16_t) * other_intrin.width * other_intrin.height);
        std::unique_ptr<uint16_t, decltype(cudaDeleterUInt16t)> out_pixels_gpu_uint16t_u_ptr(out_pixels_gpu_uint16t, cudaDeleterUInt16t);
        if (hipSuccess != cudaCallErrorStatus) {
            ROS_ERROR("GPU processing failed while allocating space for out_pixels_gpu_uint16t %s %s", hipGetErrorName(cudaCallErrorStatus), hipGetErrorString(cudaCallErrorStatus));
            return false;
        }
        cudaCallErrorStatus = hipMemcpyAsync(z_pixels_gpu, &z_pixels[0], sizeof(uint16_t) * z_intrin.width * z_intrin.height,hipMemcpyHostToDevice, myCudaStream);
        if (hipSuccess != cudaCallErrorStatus) {
            ROS_ERROR("GPU processing failed while copying z_pixels to the GPU. %s %s", hipGetErrorName(cudaCallErrorStatus), hipGetErrorString(cudaCallErrorStatus));
            return false;
        }
        align_images<<<numBlocks, threadsPerBlock, 0, myCudaStream>>>(z_intrin, z_to_other, other_intrin, z_scale, z_pixels_gpu, temporary_out_pixels_gpu);
        if (hipSuccess != hipGetLastError()) {
            ROS_ERROR("GPU processing failed while launching align_images on the GPU. %s %s", hipGetErrorName(cudaCallErrorStatus), hipGetErrorString(cudaCallErrorStatus));
            return false;
        }
        cast_to_ushort<<<numBlocks, threadsPerBlock, 0, myCudaStream>>>(out_pixels_gpu_uint16t,temporary_out_pixels_gpu, other_intrin );
        if (hipSuccess != hipGetLastError()) {
            ROS_ERROR("GPU processing failed while launching cast_to_ushort on the GPU. %s %s", hipGetErrorName(cudaCallErrorStatus), hipGetErrorString(cudaCallErrorStatus));
            return false;
        }

        cudaCallErrorStatus = hipMemcpyAsync(out_z, &out_pixels_gpu_uint16t[0], sizeof(uint16_t) * other_intrin.width * other_intrin.height,hipMemcpyDeviceToHost, myCudaStream);
        if (hipSuccess != cudaCallErrorStatus) {
            ROS_ERROR("GPU processing failed while copying out_z from the GPU. %s %s", hipGetErrorName(cudaCallErrorStatus), hipGetErrorString(cudaCallErrorStatus));
            return false;
        }

        cudaCallErrorStatus = hipStreamSynchronize(myCudaStream);
        if (hipSuccess != cudaCallErrorStatus) {
            ROS_ERROR("GPU processing failed while synchronizing streams. %s %s", hipGetErrorName(cudaCallErrorStatus), hipGetErrorString(cudaCallErrorStatus));
            return false;
        }

        // the cuda stream wrapper coming out of scope will destroy the cuda stream
        return true;
    }
}
#endif

